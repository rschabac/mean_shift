#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>

#define STB_IMAGE_IMPLEMENTATION
#define STBI_FAILURE_USERMSG
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "kdtree.h"
struct Point {
    float r, g, b, x, y;

    Point(float p1, float p2, float p3, float p4, float p5) {
        r = p1;
        g = p2;
        b = p3;
        x = p4;
        y = p5;
    }
	float* operator[](int i)  {
		switch(i) {
		case 0: return &this->r;
		case 1: return &this->g;
		case 2: return &this->b;
		case 3: return &this->x;
		case 4: return &this->y;
		default: assert(0);
		}
	}
	float distance_squared(Point *other) {
		float delta_squared = 0;
		for (int i = 5; i < 5; i++) {
			delta_squared += (*(*this)[i] - *(*other)[i]) * (*(*this)[i] - *(*other)[i]);
		}
		return delta_squared;
	}
    //somehow this makes Point be a POD type, which is important because c++ likes to do weird things
    Point() = default;
};
//simplest way to handle errors
void add_point(struct kdtree* kd, Point p) {
    assert(kd_insertf(kd, &p.r, NULL) == 0);
}
struct kdres* neighbors(struct kdtree* kd, Point p, float radius) {
    auto result = kd_nearest_rangef(kd, &p.r, radius);
    assert(result);
    return result;
}
//Still have to call neighbors before, and kd_res_free after this
#define KD_FOR(point, set) for (kd_res_itemf(set, &point.r); !kd_res_end(set); kd_res_next(set), kd_res_itemf(set, &point.r))

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

char * cpu_version(const char* image_data, int rows, int cols, float radius, float convergence_threshold) {
    char* result = (char*)malloc(rows * cols * 3);
    struct kdtree* kd = kd_create(5);
    for (int r = 0; r < rows; r++) {
        for (int c = 0; c < cols; c++) {
            const char* const base_of_pixel = &image_data[(r * rows + cols) * 3];
            add_point(kd, Point(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], r, c));
        }
    }
	
	/*
	for each point, see what it converges to
	if its convergence point is not in the map/vector, it is a new cluster
	set its r value in result to the cluster number
	Using this scheme, max clusters is 256
	
	once all are clustered, find average rgb over each cluster, then update colors appropriately
	*/
	std::vector<Point> cluster_convergences;
	cluster_convergences.reserve(256);
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			const char* const base_of_pixel = &image_data[(r * rows + cols) * 3];
			Point centroid(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], r, c);
			while(true) {
				Point new_centroid(0,0,0,0,0);
				struct kdres* near_points = neighbors(kd, centroid, radius);
				Point temp;
				KD_FOR(temp, near_points) {
					for (int i = 0; i < 5; i++) {
						*new_centroid[i] = *temp[i];
					}
				}
				int num_near_points = kd_res_size(near_points);
				kd_res_free(near_points);
				for (int i = 0; i < 5; i++) {
					*new_centroid[i] /= num_near_points;
				}
				float delta_squared = new_centroid.distance_squared(&centroid);
				centroid = new_centroid;
				if (delta_squared <= convergence_threshold * convergence_threshold) {
					break;
				}
			}
			int cluster_id = -1;
			for (int i = 0; i < cluster_convergences.size(); i++) {
				//two paths to the same center could have converged from opposite directions,
				//so using 2 * convergence_threshold here.
				if (cluster_convergences[i].distance_squared(&centroid) < 4 * convergence_threshold * convergence_threshold) {
					//this point conerges to a centroid that has already been seen before
					cluster_id = i;
					break;
				}
			}
			if (cluster_id == -1) {
				if (cluster_convergences.size() == 256) {
					fprintf(stderr, "ERROR: more than 256 clusters identified, try tweaking CONVERGENCE_THRESHOLD and SEARCH_RADIUS\n");
					assert(0);
				}
				cluster_convergences.push_back(centroid);
				cluster_id = cluster_convergences.size() - 1;
			}
			result[(r * rows + cols) * 3] = cluster_id;
		}
	}
	
	//now compute average rgb for each cluster
	const int num_clusters = cluster_convergences.size();
	std::vector<long long> average_rs(num_clusters),
							average_gs(num_clusters),
							average_bs(num_clusters);
	std::vector<int> cluster_sizes(num_clusters);
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			int cluster_num = result[(r * rows + c) * 3];
			cluster_sizes[cluster_num]++;
			const char * const base_of_pixel = &image_data[(r * rows + c) * 3];
			average_rs[cluster_num] += base_of_pixel[0];
			average_gs[cluster_num] += base_of_pixel[1];
			average_bs[cluster_num] += base_of_pixel[2];
		}
	}
	for (int i = 0; i < num_clusters; i++) {
		average_rs[i] /= cluster_sizes[i];
		average_gs[i] /= cluster_sizes[i];
		average_bs[i] /= cluster_sizes[i];
	}
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			char * const base_of_result_pixel = &result[(r * rows + c) * 3];
			int cluster_num = base_of_result_pixel[0];
			base_of_result_pixel[0] = average_rs[cluster_num];
			base_of_result_pixel[1] = average_gs[cluster_num];
			base_of_result_pixel[2] = average_bs[cluster_num];
		}
	}
	return result;
}
#define SEARCH_RADIUS 100
#define CONVERGENCE_THRESHOLD 0
int main()
{
    int rows, cols, channels;
    char* image_data = (char*)stbi_load("test_images/dapper_lad.jpg", &cols, &rows, &channels, 3);
    if (!image_data) {
        fprintf(stderr, "Error reading image: %s", stbi_failure_reason());
        return -1;
    }
    char * cpu_result = cpu_version(image_data, rows, cols, SEARCH_RADIUS, CONVERGENCE_THRESHOLD);
    stbi_write_png("output.png", cols, rows, 3, cpu_result, 0);
    free(cpu_result);
    
    stbi_image_free(image_data);
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
