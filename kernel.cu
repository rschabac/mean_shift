#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <chrono>
#include <vector>

#define STB_IMAGE_IMPLEMENTATION
#define STBI_FAILURE_USERMSG
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "kdtree.h"
struct Point {
    float r, g, b, row, col;

    __host__ __device__ Point(float p1, float p2, float p3, float p4, float p5) {
        r = p1;
        g = p2;
        b = p3;
        row = p4;
        col = p5;
    }
	__host__ __device__ float* operator[](int i)  {
		switch(i) {
		case 0: return &this->r;
		case 1: return &this->g;
		case 2: return &this->b;
		case 3: return &this->row;
		case 4: return &this->col;
		default: assert(0);
		}
	}
	__host__ __device__ float distance_squared(Point *other) {
		float delta_squared = 0;
		for (int i = 0; i < 5; i++) {
			delta_squared += (*(*this)[i] - *(*other)[i]) * (*(*this)[i] - *(*other)[i]);
		}
		return delta_squared;
	}
    //somehow this makes Point be a POD type, which is important because c++ likes to do weird things
    Point() = default;
};

//CONFIG

//#define TIME_ITERS
#define RESTRICT __restrict
//#define RESTRICT

void add_point(struct kdtree* kd, Point p) {
	//simplest way to handle errors
	auto result = kd_insertf(kd, &p.r, NULL);
	assert(result == 0);
}
struct kdres* neighbors(struct kdtree* kd, Point p, float radius) {
    auto result = kd_nearest_rangef(kd, &p.r, radius);
    assert(result);
    return result;
}
//Still have to call neighbors before, and kd_res_free after this
#define KD_FOR(point, set) for (kd_res_itemf(set, &point.r); !kd_res_end(set); kd_res_next(set), kd_res_itemf(set, &point.r))

void color_result(const unsigned char *, unsigned char *, int *, int, int, int);
unsigned char * cpu_version(const unsigned char* image_data, int rows, int cols, float radius, float convergence_threshold, bool do_color) {
    unsigned char* result = (unsigned char*)malloc(rows * cols * 3);
	int* cluster_ids = (int*)malloc(rows * cols * sizeof(int));
    struct kdtree* kd = kd_create(5);
    for (int r = 0; r < rows; r++) {
        for (int c = 0; c < cols; c++) {
            const unsigned char* const base_of_pixel = &image_data[(r * cols + c) * 3];
            add_point(kd, Point(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], r, c));
        }
    }
	
	/*
	for each point, see what it converges to
	if its convergence point is not in the map/vector, it is a new cluster
	
	once all are clustered, find average rgb over each cluster, then update colors appropriately
	*/
	std::vector<Point> cluster_convergences;
	int max_iters = 0;
	cluster_convergences.reserve(256);
	for (int r = 0; r < rows; r++) {
		//printf("now starting r = %d\n", r);
		for (int c = 0; c < cols; c++) {
			const unsigned char* const base_of_pixel = &image_data[(r * cols + c) * 3];
			Point centroid(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], r, c);
			int iters = 0;
			while(true) {
				iters++;
				Point new_centroid(0,0,0,0,0);
				struct kdres* near_points = neighbors(kd, centroid, radius);
				Point temp;
				KD_FOR(temp, near_points) {
					for (int i = 0; i < 5; i++) {
						*new_centroid[i] += *temp[i];
					}
				}
				int num_near_points = kd_res_size(near_points);
				kd_res_free(near_points);
				for (int i = 0; i < 5; i++) {
					*new_centroid[i] /= num_near_points;
				}
				float delta_squared = new_centroid.distance_squared(&centroid);
				centroid = new_centroid;
				if (delta_squared <= convergence_threshold * convergence_threshold) {
					break;
				}
			}
			if (iters > max_iters) {
				max_iters = iters;
			}
			int cluster_id = -1;
			for (int i = 0; i < cluster_convergences.size(); i++) {
				//two paths to the same center could have converged from opposite directions,
				//so using 2 * convergence_threshold here.
				if (cluster_convergences[i].distance_squared(&centroid) <= 4 * convergence_threshold * convergence_threshold) {
					//this point converges to a centroid that has already been seen before
					cluster_id = i;
					break;
				}
			}
			if (cluster_id == -1) {
				cluster_convergences.push_back(centroid);
				cluster_id = cluster_convergences.size() - 1;
			}
			cluster_ids[r * cols + c] = cluster_id;
		}
	}
	kd_free(kd);
	//fprintf(stderr, "Max iters in CPU version was %d\n", max_iters);
	
	//now compute average rgb for each cluster
	if (do_color) {
		color_result(image_data, result, cluster_ids, cluster_convergences.size(), rows, cols);
	}
	free(cluster_ids);
	return result;
}

unsigned char *cpu_version_with_trajectories(const unsigned char *image_data, int rows, int cols, float radius, float convergence_threshold, bool do_color) {
	unsigned char* result = (unsigned char*)malloc(rows * cols * 3);
	int* cluster_ids = (int*)malloc(rows * cols * sizeof(int));
    struct kdtree* source_points = kd_create(5);
    for (int r = 0; r < rows; r++) {
        for (int c = 0; c < cols; c++) {
            const unsigned char* const base_of_pixel = &image_data[(r * cols + c) * 3];
            add_point(source_points, Point(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], r, c));
        }
    }
	/*
	make a new kd tree that maps intermediate centroids to what centroid they eventually converge to
	when iterating a point, keep track of the path of the centroid in a vector
	after it has converged to centroid `k`, add all these intermediate centroids to the kd tree and have them map to (void*)`k`
	When iterating future points, if an intermediate centroid is within a certain radius of a point already in the kd tree,
	it already maps to that centroid.
	*/
	struct kdtree *endpoints = kd_create(5);
	std::vector<Point> cluster_convergences;
	cluster_convergences.reserve(256);
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			const unsigned char* const base_of_pixel = &image_data[(r * cols + c) * 3];
			Point centroid(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], r, c);
			std::vector<Point> this_trajectory;
			int cluster_id = -1;
			while (true) {
				struct kdres *traj_points = kd_nearestf(endpoints, &centroid.r);
				//traj_points can be null if endpoints is empty
				if (traj_points && kd_res_size(traj_points) != 0) {
					Point traj_point;
					int possible_cluster_id = (int)kd_res_itemf(traj_points, &traj_point.r);
					//Might be worth trying different thresholds here
					//If centroid was close enough to an already seen traj point, it will not be added to endpoints using this method.
					if (traj_point.distance_squared(&centroid) <= convergence_threshold * convergence_threshold) {
						centroid = cluster_convergences[possible_cluster_id];
						kd_res_free(traj_points);
						cluster_id = possible_cluster_id;
						break;
					}
				}
				
				Point new_centroid(0,0,0,0,0);
				struct kdres* near_points = neighbors(source_points, centroid, radius);
				Point temp;
				KD_FOR(temp, near_points) {
					for (int i = 0; i < 5; i++) {
						*new_centroid[i] += *temp[i];
					}
				}
				int num_near_points = kd_res_size(near_points);
				kd_res_free(near_points);
				for (int i = 0; i < 5; i++) {
					*new_centroid[i] /= num_near_points;
				}
				this_trajectory.push_back(new_centroid);
				float delta_squared = new_centroid.distance_squared(&centroid);
				centroid = new_centroid;
				if (delta_squared <= convergence_threshold * convergence_threshold) {
					break;
				}
			}
			if (cluster_id == -1) {
				//did not join with existing trajectory
				for (int i = 0; i < cluster_convergences.size(); i++) {
					//two paths to the same center could have converged from opposite directions,
					//so using 2 * convergence_threshold here.
					if (cluster_convergences[i].distance_squared(&centroid) <= 4 * convergence_threshold * convergence_threshold) {
						//this point converges to a centroid that has already been seen before
						cluster_id = i;
						break;
					}
				}
				if (cluster_id == -1) {
					cluster_convergences.push_back(centroid);
					cluster_id = cluster_convergences.size() - 1;
				}
			}
			cluster_ids[r * cols + c] = cluster_id;
			
			for (int i = 0; i < this_trajectory.size(); i++) {
				assert( kd_insertf(endpoints, &this_trajectory[i].r, (void*)cluster_id) == 0 );
			}
		}
	}
	kd_free(endpoints);
	kd_free(source_points);
	
	if (do_color) color_result(image_data, result, cluster_ids, cluster_convergences.size(), rows, cols);
	free(cluster_ids);
	return result;
}

void color_result(const unsigned char *image_data, unsigned char *result, int *cluster_ids, int num_clusters, int rows, int cols) {
	std::vector<long long> average_rs(num_clusters),
							average_gs(num_clusters),
							average_bs(num_clusters);
	std::vector<int> cluster_sizes(num_clusters);
	//fprintf(stderr, "%d clusters\n", num_clusters);
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			int cluster_num = cluster_ids[r * cols + c];
			cluster_sizes[cluster_num]++;
			const unsigned char * const base_of_pixel = &image_data[(r * cols + c) * 3];
			average_rs[cluster_num] += base_of_pixel[0];
			average_gs[cluster_num] += base_of_pixel[1];
			average_bs[cluster_num] += base_of_pixel[2];
		}
	}
	for (int i = 0; i < num_clusters; i++) {
		average_rs[i] /= cluster_sizes[i];
		average_gs[i] /= cluster_sizes[i];
		average_bs[i] /= cluster_sizes[i];
	}
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			char unsigned * const base_of_result_pixel = &result[(r * cols + c) * 3];
			int cluster_num = cluster_ids[r * cols + c];
			base_of_result_pixel[0] = average_rs[cluster_num];
			base_of_result_pixel[1] = average_gs[cluster_num];
			base_of_result_pixel[2] = average_bs[cluster_num];
		}
	}
}

void naive_kernel_internals(const unsigned char* RESTRICT image, int rows, int cols, int r, int c, float radius, Point* centroids, float* deltas, float convergence_threshold) {
	if (r >= rows || c >= cols) {
		return;
	}
	if (deltas[r * cols + c] <= convergence_threshold) {
		//No need to keep iterating this centroid
		return;
	}
	Point new_centroid(0, 0, 0, 0, 0);
	int num_neighbors = 0;
	Point* this_centroid = &centroids[r * cols + c];
	const float radius_squared = radius * radius;
	for (int d_r = (int)floorf(-radius) - 1; d_r <= (int)ceilf(radius) + 1; d_r++) {
		//float limit = sqrtf(radius * radius - d_r * d_r);
		//for (int d_c = floorf(-limit); d_c <= ceilf(limit); d_c++) {
		for (int d_c = (int)floorf(-radius) - 1; d_c <= (int)ceilf(radius) + 1; d_c++) {
			if (d_r * d_r + d_c * d_c > radius_squared) continue;
			int search_r = (int)floorf(this_centroid->row + d_r), search_c = (int)floorf(this_centroid->col + d_c);
			if (search_r < 0 || search_r >= rows || search_c < 0 || search_c >= cols) {
				continue;
			}
			const unsigned char* const base_of_pixel = &image[(search_r * cols + search_c) * 3];
			Point potential_neighbor(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], search_r, search_c);
			float distance_squared = potential_neighbor.distance_squared(this_centroid);
			if (distance_squared <= radius_squared) {
				num_neighbors++;
				//fprintf(stderr, "pixel at r=%d, c=%d is a neighbor of centroid r=%f, c=%f\n", search_r, search_c, this_centroid->row, this_centroid->col);
				for (int i = 0; i < 5; i++) {
					*new_centroid[i] += *potential_neighbor[i];
				}
			}
		}
	}
	for (int i = 0; i < 5; i++) {
		*new_centroid[i] /= num_neighbors;
	}
	float distance_squared = new_centroid.distance_squared(this_centroid);
	*this_centroid = new_centroid;
	/*if (deltas[r * cols + c] > distance_squared) {
		fprintf(stderr, "delta for r=%d, c=%d got bigger, was %f, now is %f\n", r, c, deltas[r * cols + c], distance_squared);
	}*/
	deltas[r * cols + c] = distance_squared;
}

template <bool EarlyStop>
__global__ void first_kernel(const unsigned char * const RESTRICT image, const int rows, const int cols, const float radius, Point * const centroids, float * const deltas, const float convergence_threshold) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x, r = blockIdx.y * blockDim.y + threadIdx.y;
	if (r >= rows || c >= cols) {
		return;
	}
	if constexpr (EarlyStop) {
		if (deltas[r * cols + c] <= convergence_threshold) {
			return;
		}
	}

	int num_neighbors = 0;
	Point *this_centroid = &centroids[r * cols + c];
	const float radius_squared = radius * radius;
	Point new_centroid(0, 0, 0, 0, 0);
	for (int d_r = (int)floorf(-radius) - 1; d_r <= (int)ceilf(radius) + 1; d_r++) {
		float limit = sqrtf(radius * radius - d_r * d_r);
		for (int d_c = floorf(-limit); d_c <= ceilf(limit); d_c++) {
		//for (int d_c = (int)floorf(-radius) - 1; d_c <= (int)ceilf(radius) + 1; d_c++) {
			if (d_r * d_r + d_c * d_c > radius_squared) continue;
			const int search_r = (int)floorf(this_centroid->row + d_r), search_c = (int)floorf(this_centroid->col + d_c);
			if (search_r < 0 || search_r >= rows || search_c < 0 || search_c >= cols) {
				continue;
			}
			const unsigned char* const base_of_pixel = &image[(search_r * cols + search_c) * 3];
			Point potential_neighbor(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], search_r, search_c);
			if (potential_neighbor.distance_squared(this_centroid) <= radius_squared) {
				num_neighbors++;
				for (int i = 0; i < 5; i++) {
					*new_centroid[i] += *potential_neighbor[i];
				}
			}
		}
	}
	for (int i = 0; i < 5; i++) {
		*new_centroid[i] /= num_neighbors;
	}
	const float distance_squared = new_centroid.distance_squared(this_centroid);
	*this_centroid = new_centroid;

	deltas[r * cols + c] = distance_squared;
}

__global__ void reg_points_kernel(const unsigned char * const RESTRICT image, const int rows, const int cols, const float radius, Point * const centroids, float * const deltas, const float convergence_threshold) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x, r = blockIdx.y * blockDim.y + threadIdx.y;
	if (r >= rows || c >= cols) {
		return;
	}
	if (deltas[r * cols + c] <= convergence_threshold) {
		return;
	}

	int num_neighbors = 0;
	Point *this_centroid = &centroids[r * cols + c];
	const float radius_squared = radius * radius;
	float new_r = 0.0f, new_g = 0.0f, new_b = 0.0f, new_row = 0.0f, new_col = 0.0f;
	const float this_centroid_row = this_centroid->row,
				this_centroid_col = this_centroid->col,
				this_centroid_r = this_centroid->r,
				this_centroid_g = this_centroid->g,
				this_centroid_b = this_centroid->b;
	for (int d_r = (int)floorf(-radius) - 1; d_r <= (int)ceilf(radius) + 1; d_r++) {
		float limit = sqrtf(radius * radius - d_r * d_r);
		for (int d_c = floorf(-limit); d_c <= ceilf(limit); d_c++) {
			if (d_r * d_r + d_c * d_c > radius_squared) continue;
			const int search_r = (int)floorf(this_centroid_row + d_r), search_c = (int)floorf(this_centroid_col + d_c);
			if (search_r < 0 || search_r >= rows || search_c < 0 || search_c >= cols) {
				continue;
			}
			const unsigned char* const base_of_pixel = &image[(search_r * cols + search_c) * 3];
			float potential_r = base_of_pixel[0],
				potential_g = base_of_pixel[1],
				potential_b = base_of_pixel[2];
			const float delta_r = potential_r - this_centroid_r;
			const float delta_g = potential_g - this_centroid_g;
			const float delta_b = potential_b - this_centroid_b;
			const float delta_row = search_r - this_centroid_row;
			const float delta_col = search_c - this_centroid_col;
			if (delta_r * delta_r + delta_g * delta_g + delta_b * delta_b + delta_row * delta_row + delta_col * delta_col <= radius_squared) {
				num_neighbors++;
				//fprintf(stderr, "pixel at r=%d, c=%d is a neighbor of centroid r=%f, c=%f\n", search_r, search_c, this_centroid->row, this_centroid->col);
				new_r += potential_r;
				new_g += potential_g;
				new_b += potential_b;
				new_row += search_r;
				new_col += search_c;
			}
		}
	}
	new_r /= num_neighbors;
	new_g /= num_neighbors;
	new_b /= num_neighbors;
	new_row /= num_neighbors;
	new_col /= num_neighbors;
	const float delta_r = new_r - this_centroid_r;
	const float delta_g = new_g - this_centroid_g;
	const float delta_b = new_b - this_centroid_b;
	const float delta_row = new_row - this_centroid_row;
	const float delta_col = new_col - this_centroid_col;
	const float distance_squared = delta_r * delta_r + delta_g * delta_g + delta_b * delta_b + delta_row * delta_row + delta_col * delta_col;
	this_centroid->r = new_r;
	this_centroid->g = new_g;
	this_centroid->b = new_b;
	this_centroid->row = new_row;
	this_centroid->col = new_col;

	deltas[r * cols + c] = distance_squared;
}

template <size_t SH_PAD>
__global__ void shmem_kernel(const unsigned char * const RESTRICT image, const int rows, const int cols, const float radius, Point * const centroids, float * const deltas, const float convergence_threshold) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x, r = blockIdx.y * blockDim.y + threadIdx.y;

	/*
	* Each block is 32x32
	* My initial guess for SEARCH_RADIUS is 50
	* can fit a max of 16384 pixels in shmem
	* Doesn't make sense for shared memory to be more than [132][132]
	* Maximum shared memory on my device is [128][128]
	* More shared mem means less L1, try varying shared mem dimensions
	* Should be at least 32x32?
	*/
	//SH_PAD is number of pixels beyond the 32x32 that should be in shared memory
	//total dimension length of shared memory
	constexpr size_t SH_DIM = 32 + 2 * SH_PAD;
	//Each thread is responsible for loading this many (squared) pixels
	constexpr size_t RESP_DIM = (SH_DIM + 31) / 32;
	__shared__ unsigned char shared[SH_DIM][SH_DIM][3];
	for (int r_offset = 0; r_offset < RESP_DIM; r_offset++) {
		for (int c_offset = 0; c_offset < RESP_DIM; c_offset++) {
			const int image_r = 32 * blockIdx.y - SH_PAD + threadIdx.y * RESP_DIM + r_offset,
				image_c = 32 * blockIdx.x - SH_PAD + threadIdx.x * RESP_DIM + c_offset;
			if (image_r < 0 || image_r >= rows || image_c < 0 || image_c >= cols) {
				continue;
			}
			const unsigned char* const base_of_pixel = &image[(image_r * cols + image_c) * 3];
			int dest_r = RESP_DIM * threadIdx.y + r_offset,
				dest_c = RESP_DIM * threadIdx.x + c_offset;
			if (dest_r >= 0 && dest_r < SH_DIM && dest_c >= 0 && dest_c < SH_DIM) {
				shared[dest_r][dest_c][0] = base_of_pixel[0];
				shared[dest_r][dest_c][1] = base_of_pixel[1];
				shared[dest_r][dest_c][2] = base_of_pixel[2];
			}
		}
	}
	__syncthreads();
	//This needs to happen after shmem is populated because out of bounds threads
	//could still responsible for some in-bounds shmem items
	if (r >= rows || c >= cols) {
		return;
	}
	if (deltas[r * cols + c] <= convergence_threshold) {
		return;
	}

	int num_neighbors = 0;
	Point *this_centroid = &centroids[r * cols + c];
	const float radius_squared = radius * radius;
	float new_r = 0.0f, new_g = 0.0f, new_b = 0.0f, new_row = 0.0f, new_col = 0.0f;
	const float this_centroid_row = this_centroid->row,
				this_centroid_col = this_centroid->col,
				this_centroid_r = this_centroid->r,
				this_centroid_g = this_centroid->g,
				this_centroid_b = this_centroid->b;
	for (int d_r = (int)floorf(-radius) - 1; d_r <= (int)ceilf(radius) + 1; d_r++) {
		//float limit = sqrtf(radius * radius - d_r * d_r);
		//for (int d_c = floorf(-limit); d_c <= ceilf(limit); d_c++) {
		for (int d_c = (int)floorf(-radius) - 1; d_c <= (int)ceilf(radius) + 1; d_c++) {
			if (d_r * d_r + d_c * d_c > radius_squared) continue;
			const int search_r = (int)floorf(this_centroid_row + d_r), search_c = (int)floorf(this_centroid_col + d_c);
			if (search_r < 0 || search_r >= rows || search_c < 0 || search_c >= cols) {
				continue;
			}
			float potential_r, potential_g, potential_b;
			/*
			if blockidx = {2,2}, shared[0][0] is image[64-SH_PAD][64-SH_PAD]
			shared[SH_PAD][SH_PAD] is image[64][64]
			shared[SH_DIM-1][SH_DIM-1] is image[96+SH_PAD-1][96+SH_PAD-1]
			sharedIdx = imgIdx - 32*blkIdx + SH_PAD
			*/
			const int shared_r = search_r - 32 * blockIdx.y + SH_PAD,
				shared_c = search_c - 32 * blockIdx.x + SH_PAD;
			if (shared_r >= 0 && shared_r < SH_DIM && shared_c >= 0 && shared_c < SH_DIM) {
				potential_r = shared[shared_r][shared_c][0];
				potential_g = shared[shared_r][shared_c][1];
				potential_b = shared[shared_r][shared_c][2];
			} else {
				const unsigned char* const base_of_pixel = &image[(search_r * cols + search_c) * 3];
				potential_r = base_of_pixel[0];
				potential_g = base_of_pixel[1];
				potential_b = base_of_pixel[2];
			}
			const float delta_r = potential_r - this_centroid_r;
			const float delta_g = potential_g - this_centroid_g;
			const float delta_b = potential_b - this_centroid_b;
			const float delta_row = search_r - this_centroid_row;
			const float delta_col = search_c - this_centroid_col;
			if (delta_r * delta_r + delta_g * delta_g + delta_b * delta_b + delta_row * delta_row + delta_col * delta_col <= radius_squared) {
				num_neighbors++;
				new_r += potential_r;
				new_g += potential_g;
				new_b += potential_b;
				new_row += search_r;
				new_col += search_c;
			}
		}
	}
	new_r /= num_neighbors;
	new_g /= num_neighbors;
	new_b /= num_neighbors;
	new_row /= num_neighbors;
	new_col /= num_neighbors;
	const float delta_r = new_r - this_centroid_r;
	const float delta_g = new_g - this_centroid_g;
	const float delta_b = new_b - this_centroid_b;
	const float delta_row = new_row - this_centroid_row;
	const float delta_col = new_col - this_centroid_col;
	float distance_squared = delta_r * delta_r + delta_g * delta_g + delta_b * delta_b + delta_row * delta_row + delta_col * delta_col;
	this_centroid->r = new_r;
	this_centroid->g = new_g;
	this_centroid->b = new_b;
	this_centroid->row = new_row;
	this_centroid->col = new_col;

	deltas[r * cols + c] = distance_squared;
}

unsigned char* sequential_gpu_version(const unsigned char* image_data, int rows, int cols, float radius, float convergence_threshold, bool do_color) {
	unsigned char *result = (unsigned char*)malloc(rows * cols * 3);
	int* cluster_ids = (int*)malloc(rows * cols * sizeof(int));
	Point *centroids = (Point*)malloc(rows * cols * sizeof(Point));
	for (int r = 0; r < rows; r++){
		for (int c = 0; c < cols; c++) {
			const unsigned char* const base_of_pixel = &image_data[(r * cols + c) * 3];
			centroids[r * cols + c] = Point(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], r, c);
		}
	}
	
	float *deltas = (float*)malloc(rows * cols * sizeof(float));
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			deltas[r * cols + c] = INFINITY;
		}
	}
	
	//int iters = 0;
	while (true) {
		for (int r = 0; r < rows; r++) {
			for (int c = 0; c < cols; c++) {
				naive_kernel_internals(image_data, rows, cols, r, c, radius, centroids, deltas, convergence_threshold);
			}
		}
		bool found_greater_than_thresh = false;
		for (int r = 0; r < rows; r++) {
			for (int c = 0; c < cols; c++) {
				if (deltas[r * cols + c] > convergence_threshold) {
					found_greater_than_thresh = true;
					break;
				}
			}
			if (found_greater_than_thresh) {
				break;
			}
		}
		if (!found_greater_than_thresh) {
			break;
		}
	}
	
	std::vector<Point> cluster_convergences;
	cluster_convergences.reserve(256);
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			int cluster_id = -1;
			for (int i = 0; i < cluster_convergences.size(); i++) {
				if (cluster_convergences[i].distance_squared(&centroids[r * cols + c]) <= 4 * convergence_threshold * convergence_threshold) {
					cluster_id = i;
					break;
				}
			}
			if (cluster_id == -1) {
				cluster_convergences.push_back(centroids[r * cols + c]);
				cluster_id = (int)cluster_convergences.size() - 1;
			}
			cluster_ids[r * cols + c] = cluster_id;
		}
	}
	if (do_color) color_result(image_data, result, cluster_ids, cluster_convergences.size(), rows, cols);
	
	free(cluster_ids);
	free(deltas);
	free(centroids);
	return result;
}

__global__ void max_in_groups(float* data, int n) {
	__shared__ float block_data[2048];
	unsigned short t_id = threadIdx.x;
	unsigned global_block_start = blockIdx.x * 2048;
	unsigned short local_offset = 2 * t_id;
	//pad out the last block's block_data with zeros
	if (global_block_start + local_offset < n) {
		block_data[local_offset] = data[global_block_start + local_offset];
		if (global_block_start + local_offset + 1 < n) {
			block_data[local_offset + 1] = data[global_block_start + local_offset + 1];
		}
		else {
			block_data[local_offset + 1] = 0.0f;
		}
	}
	else {
		block_data[local_offset] = 0.0f;
		block_data[local_offset + 1] = 0.0f;
	}

	for (unsigned short stride = 1024; stride >= 1; stride >>= 1) {
		__syncthreads();
		if (t_id < stride) {
			block_data[t_id] = max(block_data[t_id + stride], block_data[t_id]);
		}
	}

	__syncthreads();
	if (t_id == 0) {
		data[global_block_start] = block_data[0];
	}
}

__device__ int need_more_iter;

template <bool UseAtomics>
__global__ void kernel_without_deltas(const unsigned char* const RESTRICT image, const int rows, const int cols, const float radius, Point* const centroids, const float convergence_threshold) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x, r = blockIdx.y * blockDim.y + threadIdx.y;
	/*
	If a centroid has a negative .r value, return
	Compute the delta for each centroid, as before
	if it is less than the threshold, .r -= 256
	otherwise, shared_flag = true;

	__syncthreads();
	thread 0 in each block checks shared_flag
	if it is true, atomicOr(should_continue, true)
	*/
	__shared__ int need_more_shared;
	if constexpr (UseAtomics) {
		if (threadIdx.x == 0 && threadIdx.y == 0) {
			need_more_shared = 0;
		}
	}

	if (r >= rows || c >= cols) {
		return;
	}
	Point* const this_centroid = &centroids[r * cols + c];
	if (this_centroid->r < 0) {
		return;
	}
	if constexpr (UseAtomics) {
		__syncthreads();
	} else {
		need_more_shared = 0;
	}
	
	int num_neighbors = 0;
	const float radius_squared = radius * radius;
	float new_r = 0.0f, new_g = 0.0f, new_b = 0.0f, new_row = 0.0f, new_col = 0.0f;
	const float this_centroid_row = this_centroid->row,
		this_centroid_col = this_centroid->col,
		this_centroid_r = this_centroid->r,
		this_centroid_g = this_centroid->g,
		this_centroid_b = this_centroid->b;
	for (int d_r = (int)floorf(-radius) - 1; d_r <= (int)ceilf(radius) + 1; d_r++) {
		for (int d_c = (int)floorf(-radius) - 1; d_c <= (int)ceilf(radius) + 1; d_c++) {
			if (d_r * d_r + d_c * d_c > radius_squared) continue;
			const int search_r = (int)floorf(this_centroid_row + d_r), search_c = (int)floorf(this_centroid_col + d_c);
			if (search_r < 0 || search_r >= rows || search_c < 0 || search_c >= cols) {
				continue;
			}
			const unsigned char* const base_of_pixel = &image[(search_r * cols + search_c) * 3];
			const float potential_r = base_of_pixel[0],
				potential_g = base_of_pixel[1],
				potential_b = base_of_pixel[2];
			
			const float delta_r = potential_r - this_centroid_r;
			const float delta_g = potential_g - this_centroid_g;
			const float delta_b = potential_b - this_centroid_b;
			const float delta_row = search_r - this_centroid_row;
			const float delta_col = search_c - this_centroid_col;
			if (delta_r * delta_r + delta_g * delta_g + delta_b * delta_b + delta_row * delta_row + delta_col * delta_col <= radius_squared) {
				num_neighbors++;
				//fprintf(stderr, "pixel at r=%d, c=%d is a neighbor of centroid r=%f, c=%f\n", search_r, search_c, this_centroid->row, this_centroid->col);
				new_r += potential_r;
				new_g += potential_g;
				new_b += potential_b;
				new_row += search_r;
				new_col += search_c;
			}
		}
	}
	new_r /= num_neighbors;
	new_g /= num_neighbors;
	new_b /= num_neighbors;
	new_row /= num_neighbors;
	new_col /= num_neighbors;
	const float delta_r = new_r - this_centroid_r;
	const float delta_g = new_g - this_centroid_g;
	const float delta_b = new_b - this_centroid_b;
	const float delta_row = new_row - this_centroid_row;
	const float delta_col = new_col - this_centroid_col;
	float distance_squared = delta_r * delta_r + delta_g * delta_g + delta_b * delta_b + delta_row * delta_row + delta_col * delta_col;

	if (distance_squared <= convergence_threshold) {
		//this centroid has converged, do no further processing on it
		new_r -= 256;
	} else if (!need_more_shared) {
		if constexpr (UseAtomics) {
			atomicOr(&need_more_shared, 1);
		} else {
			need_more_shared = 1;
		}
	}
	this_centroid->r = new_r;
	this_centroid->g = new_g;
	this_centroid->b = new_b;
	this_centroid->row = new_row;
	this_centroid->col = new_col;

	/*
	Only one thread in each block needs to set the global flag
	Can't always choose thread (0,0) in the block, this may have already returned
	*/
	if (need_more_shared) {
		if constexpr (UseAtomics) {
			atomicOr(&need_more_iter, 1);
		} else {
			need_more_iter = 1;
		}
	}
}

enum KernelType {
	First,
	RegPoints,
	Shmem,
	NoDeltas
};

template <KernelType WhichKernel, bool EarlyStop = true, size_t SH_PAD = 48, bool UseAtomics = true>
unsigned char * GPU_driver(const unsigned char *image_data, int rows, int cols, float radius, float convergence_threshold, bool do_color) {
	unsigned char *result = (unsigned char*)malloc(rows * cols * 3);
	int* cluster_ids = (int*)malloc(rows * cols * sizeof(int));
	hipError_t err_code = hipSuccess;
	err_code = hipSetDevice(0);
	Point *host_centroids = (Point*)malloc(rows * cols * sizeof(Point));
	for (int r = 0; r < rows; r++){
		for (int c = 0; c < cols; c++) {
			const unsigned char* const base_of_pixel = &image_data[(r * cols + c) * 3];
			host_centroids[r * cols + c] = Point(base_of_pixel[0], base_of_pixel[1], base_of_pixel[2], r, c);
		}
	}
	Point *dev_centroids = NULL;
	err_code = hipMalloc((void**)&dev_centroids, rows * cols * sizeof(Point));
	err_code = hipMemcpy(dev_centroids, host_centroids, rows * cols * sizeof(Point), hipMemcpyHostToDevice);
	
	float *host_deltas = NULL, *dev_deltas = NULL;
	if constexpr (WhichKernel != NoDeltas) {
		host_deltas = (float*)malloc(rows * cols * sizeof(float));
		for (int i = 0; i < rows * cols; i++) {
			host_deltas[i] = INFINITY;
		}
		dev_deltas = NULL;
		err_code = hipMalloc((void**)&dev_deltas, rows * cols * sizeof(float));
		err_code = hipMemcpy(dev_deltas, host_deltas, rows * cols * sizeof(float), hipMemcpyHostToDevice);
	}
	
	
	unsigned char *dev_image = NULL;
	err_code = hipMalloc((void**)&dev_image, rows * cols * 3);
	err_code = hipMemcpy(dev_image, image_data, rows * cols * 3, hipMemcpyHostToDevice);
	
#ifdef TIME_ITERS
	std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
#endif
	for (int iters = 0; ; iters++) {
#ifdef TIME_ITERS
		start = std::chrono::high_resolution_clock::now();
#endif
		//fprintf(stderr, "now starting iter %d\n", iters++);
		//My device (NVIDIA GeForce GTX 1660) has a max of 1024 threads per block
		dim3 block_dims(32, 32);
		dim3 grid_dims((cols + 31)/32, (rows + 31)/32);
		if constexpr (WhichKernel == First) {
			first_kernel<EarlyStop><<<grid_dims, block_dims>>>(dev_image, rows, cols, radius, dev_centroids, dev_deltas, convergence_threshold);
		} else if constexpr (WhichKernel == RegPoints) {
			reg_points_kernel<<<grid_dims, block_dims>>>(dev_image, rows, cols, radius, dev_centroids, dev_deltas, convergence_threshold);
		} else if constexpr (WhichKernel == Shmem) {
			shmem_kernel<SH_PAD><<<grid_dims, block_dims>>>(dev_image, rows, cols, radius, dev_centroids, dev_deltas, convergence_threshold);
		} else if constexpr (WhichKernel == NoDeltas) {
			int zero = 0;
			err_code = hipMemcpyToSymbol(HIP_SYMBOL(need_more_iter), &zero, sizeof(int), 0, hipMemcpyHostToDevice);
			kernel_without_deltas<UseAtomics><<<grid_dims, block_dims>>>(dev_image, rows, cols, radius, dev_centroids, convergence_threshold);
		} else {
			fprintf(stderr, "Invalid Kernel specified\n");
			abort();
		}
		
#ifdef _DEBUG
		err_code = hipGetLastError(); //errors from launching the kernel
		err_code = hipDeviceSynchronize(); //errors that happened during the kernel launch
#endif
		if constexpr (WhichKernel != NoDeltas) {
			int blocks_necessary = (int)ceil((float)(rows * cols) / 2048.0f); //1024 is max threads per block on my device
			max_in_groups<<<blocks_necessary, 1024>>>(dev_deltas, rows * cols);
#ifdef _DEBUG
			err_code = hipGetLastError();
			err_code = hipDeviceSynchronize();
#endif
			bool all_below_threshold = true;
			//Maybe it would be faster to make one big array here and just do one memcpy into that
			for (int i = 0; i < rows * cols; i += 2048) {
				float temp;
				hipMemcpy(&temp, &dev_deltas[i], sizeof(float), hipMemcpyDeviceToHost);
				if (temp > convergence_threshold) {
					all_below_threshold = false;
					break;
				}
			}
#ifdef TIME_ITERS
			end = std::chrono::high_resolution_clock::now();
			printf("iter %d took %5ld ms\n", iters, std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());
#endif
			if (all_below_threshold) {
				break;
			}
		} else { //using a NoDeltas kernel, stopping condition is different
			int should_continue_host = 1;
			err_code = hipMemcpyFromSymbol(&should_continue_host, HIP_SYMBOL(need_more_iter), sizeof(int), 0, hipMemcpyDeviceToHost);
			if (!should_continue_host) {
				break;
			}
		}
		
	}
	
	err_code = hipMemcpy(host_centroids, dev_centroids, rows * cols * sizeof(Point), hipMemcpyDeviceToHost);
	
	std::vector<Point> cluster_convergences;
	cluster_convergences.reserve(256);
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < cols; c++) {
			int cluster_id = -1;
			for (int i = 0; i < cluster_convergences.size(); i++) {
				if (cluster_convergences[i].distance_squared(&host_centroids[r * cols + c]) <= 4 * convergence_threshold * convergence_threshold) {
					cluster_id = i;
					break;
				}
			}
			if (cluster_id == -1) {
				cluster_convergences.push_back(host_centroids[r * cols + c]);
				cluster_id = cluster_convergences.size() - 1;
			}
			cluster_ids[r * cols + c] = cluster_id;
		}
	}
	if (do_color) color_result(image_data, result, cluster_ids, cluster_convergences.size(), rows, cols);
	
	free(host_centroids);
	hipFree(dev_centroids);
	free(host_deltas);
	hipFree(dev_deltas);
	hipFree(dev_image);
	(void)err_code;
	return result;
	
}

void timings(const char* filename, float radius, float convergence_threshold) {
	hipFree(0); //Force init CUDA runtime
	printf("Now timing on %s with radius %f and threshold %f\n", filename, radius, convergence_threshold);
	int rows, cols, channels;
	unsigned char* image_data = (unsigned char*)stbi_load(filename, &cols, &rows, &channels, 3);
	if (!image_data) {
		fprintf(stderr, "Error reading image: %s\n", stbi_failure_reason());
		return;
	}

	std::chrono::time_point < std::chrono::high_resolution_clock > start, end;
#define TIME(name, stmt) do { \
	start = std::chrono::high_resolution_clock::now(); \
	stmt; \
	end = std::chrono::high_resolution_clock::now(); \
	printf(name ": %10lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()); \
} while (0)
	
	//TIME("naive CPU            ", cpu_version(image_data, rows, cols, radius, convergence_threshold, false));
	//TIME("CPU with trajectories", cpu_version_with_trajectories(image_data, rows, cols, radius, convergence_threshold, false));
	//TIME("first kernel         ", (GPU_driver<First, false, 0, false>(image_data, rows, cols, radius, convergence_threshold, false)));
	//TIME("with early stop      ", (GPU_driver<First, true, 0, false>(image_data, rows, cols, radius, convergence_threshold, false)));
	TIME("points in regs       ", GPU_driver<RegPoints>(image_data, rows, cols, radius, convergence_threshold, false));
	TIME("Shmem, pad=48        ", (GPU_driver<Shmem, false, 48, false>(image_data, rows, cols, radius, convergence_threshold, false)));
	//TIME("Shmem, pad=32        ", (GPU_driver<Shmem, false, 32, false>(image_data, rows, cols, radius, convergence_threshold, false)));
	//TIME("Shmem, pad=16        ", (GPU_driver<Shmem, false, 16, false>(image_data, rows, cols, radius, convergence_threshold, false)));
	//TIME("Shmem, pad=8         ", (GPU_driver<Shmem, false, 8, false>(image_data, rows, cols, radius, convergence_threshold, false)));
	TIME("no deltas            ", (GPU_driver<NoDeltas, false, 0, false>(image_data, rows, cols, radius, convergence_threshold, false)));
	TIME("no deltas, no atomics", GPU_driver<NoDeltas>(image_data, rows, cols, radius, convergence_threshold, false));
}
int main()
{
	//timings("test_images/dapper_lad_smaller.jpg", 50, 10);
	//timings("test_images/dapper_lad.jpg", 50, 10);
	timings("test_images/campus.jpg", 10, 10);
	timings("test_images/campus.jpg", 50, 10);
	timings("test_images/campus.jpg", 100, 10);
	
	//timings("test_images/eas_1500x100.jpg", 50, 50);
	return;

    int rows, cols, channels;
    unsigned char* image_data = (unsigned char*)stbi_load("test_images/dapper_lad_smaller.jpg", &cols, &rows, &channels, 3);
    if (!image_data) {
        fprintf(stderr, "Error reading image: %s\n", stbi_failure_reason());
        return -1;
    }
#define WRITE_IMG(filename, stmt) do { \
	unsigned char * result = stmt; \
	stbi_write_png(filename, cols, rows, 3, result, 0); \
	free(result); \
} while (0)
	const float radius = 50, convergence_threshold = 10;
	WRITE_IMG("cpu_output.png", cpu_version(image_data, rows, cols, radius, convergence_threshold, true));
	WRITE_IMG("cpu_traj_output.png", cpu_version_with_trajectories(image_data, rows, cols, radius, convergence_threshold, true));

	WRITE_IMG("first_kernel_output.png", (GPU_driver<First, false, 0, false>(image_data, rows, cols, radius, convergence_threshold, true)));
	WRITE_IMG("early_stop_output.png", (GPU_driver<First, true, 0, false>(image_data, rows, cols, radius, convergence_threshold, true)));
	WRITE_IMG("reg_points_output.png", GPU_driver<RegPoints>(image_data, rows, cols, radius, convergence_threshold, true));
	WRITE_IMG("shmem_pad_48_output.png", (GPU_driver<Shmem, false, 48, false>(image_data, rows, cols, radius, convergence_threshold, true)));
	WRITE_IMG("Shmem_pad_32_output.png", (GPU_driver<Shmem, false, 32, false>(image_data, rows, cols, radius, convergence_threshold, true)));
	WRITE_IMG("Shmem_pad_16_output.png", (GPU_driver<Shmem, false, 16, false>(image_data, rows, cols, radius, convergence_threshold, true)));
	WRITE_IMG("Shmem_pad_8_output.png", (GPU_driver<Shmem, false, 8, false>(image_data, rows, cols, radius, convergence_threshold, true)));
	WRITE_IMG("no_deltas_output.png", (GPU_driver<NoDeltas, false, 0, false>(image_data, rows, cols, radius, convergence_threshold, true)));
	WRITE_IMG("no_atomics_output.png", GPU_driver<NoDeltas>(image_data, rows, cols, radius, convergence_threshold, true));
	
	stbi_image_free(image_data);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!\n");
        return 1;
    }

    return 0;
}